#include "hip/hip_runtime.h"
/* Simple example demonstrating how to use MPI with CUDA
*
*  Generate some random numbers on one node.
*  Dispatch them to all nodes.
*  Compute their square root on each node's GPU.
*  Compute the average of the results using MPI.
*
*/

#include <iostream>
#include "vector_add.h"

using std::cerr;
using std::endl;


#define CUDA_CHECK(call) \
    if((call) != hipSuccess) { \
        hipError_t err = hipGetLastError(); \
        cerr << "CUDA error calling \""#call"\", code is " << err << endl; \
    }

__global__ void vectorAddBin(float *Md, float *Nd, int *Pd, int width)
{
  int tid;

  tid = blockIdx.x * blockDim.x + threadIdx.x;

  while (tid < width) {
     Pd[tid] = (Md[tid] + Nd[tid] + 20) * 2;
     tid += blockDim.x * gridDim.x;
  }
  return;
}

void computeAddGPU(float *vector_a, float *vector_b, int *vector_res, int size)
{
   float *d_a;
   float *d_b;
   int *d_res;

   int blockSize = 1024;
   int gridSize = (int)ceil((float)size/blockSize);

   CUDA_CHECK(hipMalloc(&d_a, sizeof(float)*size));
   CUDA_CHECK(hipMalloc(&d_b, sizeof(float)*size));
   CUDA_CHECK(hipMalloc(&d_res, sizeof(int)*size));

   CUDA_CHECK(hipMemcpy(d_a, vector_a, sizeof(float)*size, hipMemcpyHostToDevice));
   CUDA_CHECK(hipMemcpy(d_b, vector_b, sizeof(float)*size, hipMemcpyHostToDevice));

   vectorAddBin<<<gridSize, blockSize>>>(d_a, d_b, d_res, size);

   CUDA_CHECK(hipMemcpy(vector_res, d_res, sizeof(int)*size, hipMemcpyDeviceToHost));

   CUDA_CHECK(hipFree(d_a));
   CUDA_CHECK(hipFree(d_b));
   CUDA_CHECK(hipFree(d_res));
}
